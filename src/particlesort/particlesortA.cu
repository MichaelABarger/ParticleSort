#include "hip/hip_runtime.h"
#ifndef PARTICLE_SORT_CU
#define PARTICLE_SORT_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../testharness/testharness.h"

#define LIFESPAN 50


enum particleState { ALIVE, DYING, DEAD };


__device__ unsigned short *global_mem;
__device__ int idle_ct = 0;

__device__ void Collide (signed char *, char *, enum particleState *);
__device__ void Pass (char *, enum particleState *);
__device__ void Revive (signed char *, char *, enum particleState *, const signed char *);
__device__ void Die (enum particleState *);



#define COLLIDE Collide(&velocity,&momentum,&state)
#define PASS Pass(&momentum,&state)
#define REVIVE Revive(&velocity,&momentum,&state,&direction)
#define DIE Die(&state)
__global__ void ParticleSort (unsigned short *global_mem, unsigned long size)
{
	extern __shared__ int shmem [];
	int *end = &shmem[size - 1];
	int threadID = blockDim.x * blockIdx.x + threadIdx.x;

	signed char direction = -1;

	/* particle initialization */
	enum particleState state = ALIVE;
	signed char velocity = 1 - (threadID & 0x01) * 2;
	int *position = shmem + threadID;
	unsigned int value = global_mem[threadID];
	char momentum = LIFESPAN;

	/* slot initialization */
	int *here = position;

	/* main sorting loop */
	do {
		/* prepare */
		*here = 0;
		direction = -direction;
		__syncthreads();

		/* move position if velocity is same as direction */
		/* perform wall collisions */
		if (state != DEAD && velocity == direction) {
			position += velocity;
			if (position < shmem) {
				position = shmem;
				COLLIDE;
			} else if (position > end) {
				position = end;
				COLLIDE;
			}
		}

		/* resolve collisions */
		*position += ((state != DEAD) ? velocity : -direction) * value;
		__syncthreads();
		
		int sum = *position;
		int abs_sum = abs(sum);

		switch (state) {

		case DEAD:
			if (abs_sum < value)
				REVIVE;
			break;

		case DYING:
			if (abs_sum == value) {
				DIE;
				break;
			} /* fall through if not */

		case ALIVE:
			if ((abs_sum == value) || (sum == 0))
				break;
			if (sum < 0) 
				COLLIDE;
			else 
				PASS;
		}
		__syncthreads();

	} while (idle_ct < size);

	/* we're done. copy everything back into global memory */
	*position = value;
	__syncthreads();
	global_mem[threadID] = *here;
}

__device__ void Collide (signed char *velocity, char *momentum, enum particleState *state)
{
	*velocity = -(*velocity);
	if (--(*momentum) <= 0)
		*state = DYING;
}

__device__ void Pass (char *momentum, enum particleState *state)
{
	*momentum = min(*momentum + 1, LIFESPAN);
	*state = ALIVE;
}

__device__ void Revive (signed char *velocity, char *momentum, enum particleState *state, const signed char *direction)
{
	*momentum = 1;
	*velocity = -(*direction);
	*state = ALIVE;
	atomicAdd(&idle_ct, -1);
}

__device__ void Die (enum particleState *state)
{
	*state = DEAD;
	atomicAdd(&idle_ct, 1);
}



void ErrorCheck (hipError_t cerr, const char *str)
{
	if (cerr == hipSuccess) 
		return;
	fprintf(stderr, "CUDA Runtime Error: %s\n at %s\n", hipGetErrorString(cerr), str);
	exit(-1);
}


extern void sort (unsigned short *buffer, unsigned long size)
{
	ErrorCheck(hipMalloc(&global_mem, size * sizeof(unsigned short)), "hipMalloc global");
	
	ErrorCheck(hipMemcpy(global_mem, buffer, size * sizeof(unsigned short), hipMemcpyHostToDevice), "hipMemcpy host->device global");

	dim3 grid (1, 1, 1);
	dim3 block (size, 1, 1);
	size_t shmem_size = size * 4;
	ParticleSort<<<grid, block, shmem_size>>>(global_mem, size);

	ErrorCheck(hipMemcpy(buffer, global_mem, size * 2, hipMemcpyDeviceToHost), "hipMemcpy device->host");

	ErrorCheck(hipFree(global_mem), "hipFree global");
}

int main (int argc, char **argv)
{
	unsigned long elapsed = TestHarness(sort);
	fprintf(stderr, "Sort complete; time elapsed: %lu ms\n", elapsed);
	exit(EXIT__SUCCESS);
}

#endif
