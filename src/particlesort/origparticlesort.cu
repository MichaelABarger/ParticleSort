#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "testharness.h"

__device__ unsigned short *global_mem;


__global__ void ParticleSort (unsigned short *global_mem, unsigned long total_size)
{
	/* REGISTER INITIALIZATION */
	extern __shared__ int shared [];
	int threadID = blockDim.x * blockIdx.x + threadIdx.x;

	unsigned short particle_value = global_mem[threadID];
	int *particle_position, *slot;
        particle_position = slot = shared + 1 + threadID;
	float particle_velocity = 8.0f - (particle_value & 0x0001) * 16.0f;

	int is_idle = 0;


	/* SHARED MEMORY INITIALIZATION */
	if (threadID == 0)
		*shared = 0;

	/* MAIN LOOP */
	do {
		/* set slot value to 0 */
		*slot = 0;

		/* move non-idle particles */
		if (!is_idle) {
			if (abs(particle_velocity) < 0.5f) { /* idle particles that are too slow */
				/* (void) atomicAdd(shared, 1);*/
				(*shared)++;
				particle_velocity = 1.0f;
				is_idle = 1;
			} else { 
				/* move particles that still have velocity */
				particle_position += __float2int_rn(particle_velocity);
				if (particle_position < (shared + 1)) {
					particle_position = shared + 1;
					particle_velocity *= -0.9f;	
				} else if (particle_position > (shared + 2 * blockDim.x + 1)) {
					particle_position = shared + 2 * blockDim.x + 1;
					particle_velocity *= -0.9f;	
				}
			}
		}

		/* add particle's value to current-position slot's running sum */
		/* this happens whether idle or not */
		*particle_position += copysignf(particle_value, particle_velocity);
		/*(void) atomicAdd(particle_position, copysignf(particle_value, particle_velocity));*/
		__syncthreads();

		/* do collisions */
		if ((signbit(*particle_position) != signbit(particle_velocity)) || (abs(*particle_position) < abs(particle_value))) {
			if (is_idle) {
				(*shared)--;
				is_idle = 0;
			}
			particle_velocity = copysignf(particle_velocity, particle_velocity) * 0.9f;
		}
/*
		else if (is_idle && (*particle_position < 0)) {
			(*shared)--;
			/*(void) atomicSub(shared, 1);*//*
			is_idle = 0;
			particle_velocity = 
		}
		*/
		__syncthreads();
	} while (*shared < blockDim.x);

	/* END OF LIFE CLEAN-UP */
	*particle_position = particle_value;
	__syncthreads();
	global_mem[threadID] = *slot;
}

__global__ void ParticleSort2(unsigned short *global_mem, unsigned long total_size)
{
	global_mem[blockDim.x * blockIdx.x + threadIdx.x] = 5;
}

void ErrorCheck (hipError_t cerr, const char *str)
{
	if (cerr == hipSuccess) 
		return;
	fprintf(stderr, "CUDA Runtime Error: %s\n at %s\n", hipGetErrorString(cerr), str);
	exit(-1);
}


extern void sort (unsigned short *buffer, unsigned long size)
{
	ErrorCheck(hipMalloc(&global_mem, size * 2), "hipMalloc");
	
	ErrorCheck(hipMemcpy(global_mem, buffer, size * 2, hipMemcpyHostToDevice), "hipMemcpy host->device");

	dim3 grid (1, 1, 1);
	dim3 block (size, 1, 1);
	size_t shm_size = (size + 1) * 4;
	ParticleSort<<<grid, block, shm_size>>>(global_mem, size);
	ErrorCheck(hipGetLastError(), "kernel execution");


	ErrorCheck(hipMemcpy(buffer, global_mem, size * 2, hipMemcpyDeviceToHost), "hipMemcpy device->host");
	ErrorCheck(hipFree(global_mem), "hipFree");
}

int main (int argc, char **argv)
{
	unsigned long elapsed = TestHarness(sort);
	fprintf(stderr, "Sort complete; time elapsed: %lu ms\n", elapsed);
	exit(EXIT__SUCCESS);
}
