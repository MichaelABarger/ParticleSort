#include "hip/hip_runtime.h"
/**
 * particlesort.cu
 * GP-GPU CUDA implementation of ParticleSort
 * implementation by Michael Barger (bargerm@cs.pdx.edu)
 * algorithm by Max Orhai
 * under mentorship of Professor Black, Portland State U
 * November, 2012
 */


#ifndef PARTICLESORT_CU
#define PARTICLESORT_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../testharness/testharness.h"

#define BLOCK 512
#define BUFFER_SIZE 16
#define MAX_MOMENTUM 0xF
#define MOMENTUM_INIT 0xF0000000
#define MOMENTUM_WIDTH 4
#define COLOR_WIDTH 32 - MOMENTUM_WIDTH
#define COLOR_MASK 0x0fffffff
#define TRUE 1
#define BOOST 1
#define ENTROPY 1
#define FALSE 0
#define INCREASE_MOMENTUM(p) ((p).momentum=min((p).momentum+BOOST,MAX_MOMENTUM))
#define INCREASE_MOMENTUM_PTR(p) ((p)->momentum=min((p)->momentum+BOOST,MAX_MOMENTUM))
#define DECREASE_MOMENTUM(p) ((p).momentum=max((p).momentum-ENTROPY,0))
#define DECREASE_MOMENTUM_PTR(p) ((p)->momentum=max((p)->momentum-ENTROPY,0))
#define RESET(p) (p).color=0;(p).momentum=0


/// CUDA DEVICE KERNEL ////////////////////////////////////////////////////////////////////
struct particle {
	unsigned int color;
	unsigned char momentum;
};

static __device__ void ReadParticle (const unsigned int, struct particle *);
static __device__ void WriteParticle (const struct particle *, volatile unsigned int *);
static __device__ void Collide (struct particle *, struct particle *);
static __device__ void Bump (struct particle *, unsigned int *);
static __device__ void Reside (struct particle *, unsigned int *);
static __device__ void Swap (struct particle *, struct particle *);

extern "C" __global__ void ParticleSort (unsigned int *global_mem,
					 unsigned int *transblock_buffers,
					 unsigned int *buffer_flags,
					 unsigned long size)
{
	/* define shared memory */
	volatile __shared__ unsigned int beginning [BLOCK];
	__shared__ unsigned int left_incoming [BUFFER_SIZE];
	__shared__ unsigned int left_outgoing [BUFFER_SIZE];
	__shared__ unsigned int right_incoming [BUFFER_SIZE];
	__shared__ unsigned int right_outgoing [BUFFER_SIZE];
	__shared__ unsigned int *cur_left_incoming, *cur_left_outgoing,
		 			 *cur_right_incoming, *cur_right_outgoing;
	__shared__ unsigned int isNotComplete;


	/* define registers */
	const int absThreadID = blockIdx.x * blockDim.x + threadIdx.x;
	
	struct particle going_left, going_right;

	enum {BEGINNING, LEFT, MIDDLE, RIGHT, END, IDLE} role;
	if (absThreadID == 0) role = BEGINNING;
	else if (absThreadID == size - 1) role = END;
	else if (absThreadID >= size) role = IDLE;
	else if (threadIdx.x == 0) role = LEFT;
	else if (threadIdx.x == blockDim.x - 1) role = RIGHT;
	else role = MIDDLE;

	volatile unsigned int *const here = beginning + threadIdx.x;

	unsigned int resident;
	signed char i = 0;


	/* initial coalesced global memory read */
	if (role != IDLE) {
		resident = MOMENTUM_INIT | (global_mem[absThreadID] + 1);
		if (threadIdx.x & 0x01 || role == END) {
			ReadParticle(resident, &going_left);
			resident = 0;
		}
	}
	switch (role) {
	case BEGINNING:
		*here = 0;
		// fall through
	case MIDDLE:
		*(here + 1) = resident;
	}
	resident = 0;
	__syncthreads();



	/* sorting loop */
	do {
		if (role == BEGINNING)
			isNotComplete = FALSE;

		// non-diverging conditional
		if (i & 0x01) { // if moving left
			if (role != IDLE) {
				ReadParticle(*here, &going_left);

				if (going_left.color) {
					if (going_right.color)
						Collide(&going_left, &going_right);
					if (resident) {
						if (going_left.color > resident)
							Bump(&going_left, &resident);
					} else {
						if (!going_right.color && !going_left.momentum)
							Reside(&going_left, &resident);
					}
				}
			}
			__syncthreads();
			// prepare for moving right
			switch (role) {
			case BEGINNING:
				if (going_left.color)
					DECREASE_MOMENTUM(going_left);
				WriteParticle(&going_left, here);
				RESET(going_left);
				// fall through
			case MIDDLE:
				WriteParticle(&going_right, here + 1);
			}
		} else { // if moving right
			if (role != IDLE) {
				ReadParticle(*here, &going_right);

				if (going_right.color) {
					if (going_left.color)
						Collide(&going_left, &going_right);
					if (resident) {
						if (going_right.color < resident)
							Bump(&going_right, &resident);
					} else {
						if (!going_left.color && !going_right.momentum)
							Reside(&going_right, &resident);
					}
				}
			}
			__syncthreads();
			// prepare for moving left
			switch (role) {
			case END:
				if (going_right.color)
					DECREASE_MOMENTUM(going_right);
				WriteParticle(&going_right, here);
				RESET(going_right);
				// fall through
			case MIDDLE:
				WriteParticle(&going_left, here - 1);
			}
		}
		++i;
		if ((role != IDLE) && !resident)
			isNotComplete = TRUE;
		__syncthreads();
	} while (isNotComplete);

	/* read sorted values back to array */
	if (role != IDLE)
		global_mem[absThreadID] = ((resident - 1) & COLOR_MASK);
}

static __device__ void ReadParticle (const unsigned int src, struct particle *dest)
{
	dest->momentum = src >> COLOR_WIDTH;
	dest->color = src & COLOR_MASK;
}

static __device__ void WriteParticle (const struct particle *src, volatile unsigned int *dest)
{
	*dest = (src->momentum << COLOR_WIDTH) | src->color;
}

static __device__ void Collide (struct particle *L, struct particle *R)
{
	if (L->color < R->color) {
		INCREASE_MOMENTUM_PTR(L);
		INCREASE_MOMENTUM_PTR(R);
	} else {
		DECREASE_MOMENTUM_PTR(L);
		DECREASE_MOMENTUM_PTR(R);
		Swap(L, R);
	}
}

static __device__ void Bump (struct particle *incoming, unsigned int *resident)
{
	unsigned int temp = incoming->color;
	incoming->color = *resident;
	DECREASE_MOMENTUM_PTR(incoming);
	*resident = temp;
}

static __device__ void Reside (struct particle *incoming, unsigned int *resident)
{
	*resident = incoming->color;
	incoming->color = 0;
}

static __device__ void Swap (struct particle *L, struct particle *R)
{
		L->color ^= R->color;
		R->color ^= L->color;
		L->color ^= R->color;
		L->momentum ^= R->momentum;
		R->momentum ^= L->momentum;
		L->momentum ^= R->momentum;
}

/// CUDA HOST /////////////////////////////////////////////////////////////////////////////
static void ErrorCheck (hipError_t cerr, const char *str);
__device__ unsigned int *global_mem;
__device__ unsigned int *transblock_buffers;
__device__ unsigned int *buffer_flags;

extern "C" void sort (unsigned int *buffer, unsigned long size)
{
	dim3 grid (1);
	dim3 block (BLOCK);
	size_t global_mem_size = size * sizeof(int);
	size_t transblock_buffers_size = BUFFER_SIZE * (block.x - 1) * sizeof(int);
	size_t buffer_flags_size = (block.x - 1) * sizeof(int) * 2;

	ErrorCheck(hipMalloc(&global_mem, global_mem_size), "hipMalloc global");
	ErrorCheck(hipMemcpy(global_mem, buffer, global_mem_size, hipMemcpyHostToDevice),
			"hipMemcpy device->host");

	ErrorCheck(hipMalloc(&transblock_buffers, transblock_buffers_size), "hipMalloc buffers");
	ErrorCheck(hipMemset(transblock_buffers, 0, transblock_buffers_size), "hipMemset buffers");
	ErrorCheck(hipMalloc(&buffer_flags, buffer_flags_size), "hipMalloc buffer-flags");
	ErrorCheck(hipMemset(buffer_flags, 0, buffer_flags_size), "hipMemset buffer-flags");

	ParticleSort<<<grid, block>>>(global_mem, transblock_buffers, buffer_flags, size);
	ErrorCheck(hipMemcpy(buffer, global_mem, global_mem_size, hipMemcpyDeviceToHost),
			"hipMemcpy host->device");

	ErrorCheck(hipFree(global_mem), "hipFree global");
	ErrorCheck(hipFree(transblock_buffers), "hipFree buffers");
	ErrorCheck(hipFree(buffer_flags), "hipFree buffer-flags");


}

static void ErrorCheck (hipError_t cerr, const char *str)
{
	if (cerr == hipSuccess) return;
	fprintf(stderr, "CUDA Runtime Error: %s\n at %s\n", hipGetErrorString(cerr), str);
	exit(EXIT_FAILURE);
}

/// MAIN //////////////////////////////////////////////////////////////////////////////////
int main (int argc, char **argv)
{
	unsigned long elapsed = TestHarness(sort);
	fprintf(stderr, "Sort complete; time elapsed: %lu ms\n", elapsed);
	exit(EXIT_SUCCESS);
}

#endif
