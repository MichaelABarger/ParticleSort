#include "hip/hip_runtime.h"
/**
 * particlesort.cu
 * GP-GPU CUDA implementation of ParticleSort
 * implementation by Michael Barger (bargerm@cs.pdx.edu)
 * algorithm by Max Orhai
 * under mentorship of Professor Black, Portland State U
 * November, 2012
 */


#ifndef PARTICLESORT_CU
#define PARTICLESORT_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../testharness/testharness.h"

#define BLOCK 512
#define MAX_MOMENTUM 0xF
#define MOMENTUM_INIT 0xF0000000
#define MOMENTUM_WIDTH 4
#define COLOR_WIDTH 32 - MOMENTUM_WIDTH
#define COLOR_MASK 0x0fffffff
#define TRUE 1
#define BOOST 1
#define ENTROPY 1
#define FALSE 0
#define INCREASE_MOMENTUM(p) ((p).momentum=min((p).momentum+BOOST,MAX_MOMENTUM))
#define INCREASE_MOMENTUM_PTR(p) ((p)->momentum=min((p)->momentum+BOOST,MAX_MOMENTUM))
#define DECREASE_MOMENTUM(p) ((p).momentum=max((p).momentum-ENTROPY,0))
#define DECREASE_MOMENTUM_PTR(p) ((p)->momentum=max((p)->momentum-ENTROPY,0))
#define RESET(p) (p).color=0;(p).momentum=0


/// CUDA DEVICE KERNEL ////////////////////////////////////////////////////////////////////
struct particle {
	unsigned int color;
	unsigned char momentum;
};

static __device__ void ReadParticle (const unsigned int, struct particle *);
static __device__ void WriteParticle (const struct particle *, volatile unsigned int *);
static __device__ void Collide (struct particle *, struct particle *);
static __device__ void Bump (struct particle *, unsigned int *);
static __device__ void Reside (struct particle *, unsigned int *);

extern "C" __global__ void ParticleSort (unsigned int *global_mem,
					 unsigned long size)
{
	/* define shared memory */
	volatile __shared__ unsigned int beginning [BLOCK];
	volatile __shared__ unsigned int isNotComplete;


	/* define registers */
	const int absThreadID = blockIdx.x * blockDim.x + threadIdx.x;
	const int end = min(blockDim.x - 1, (int)size - 1);
	
	struct particle going_left, going_right;

	enum {BEGINNING, MIDDLE, END} role;
	if (threadIdx.x == 0) role = BEGINNING;
	else if (threadIdx.x == end) role = END;
	else role = MIDDLE;

	volatile unsigned int *const here = beginning + threadIdx.x;

	unsigned int resident;
	signed char i = 0;




	/* initial coalesced global memory read */
	resident = MOMENTUM_INIT | (global_mem[absThreadID] + 1);
	if (threadIdx.x & 0x01) {
		ReadParticle(resident, &going_left);
		resident = 0;
	}
	switch (role) {
	case BEGINNING:
		*here = 0;
		// fall through
	case MIDDLE:
		*(here + 1) = resident;
	}
	resident = 0;



	/* sorting loop */
	do {
		if (threadIdx.x == 0)
			isNotComplete = FALSE;

		// non-diverging conditional
		if (i++ & 0x01) { // if moving left

			ReadParticle(*here, &going_left);

			if (going_left.color) {
				if (going_right.color)
					Collide(&going_left, &going_right);
				if (resident && (going_left.color < resident))
					Bump(&going_left, &resident);
				else if (!resident && !going_right.color && !going_left.momentum)
					Reside(&going_left, &resident);
			}

			__syncthreads();
			// prepare for moving right
			switch (role) {
			case BEGINNING:
				if (going_left.color)
					DECREASE_MOMENTUM(going_left);
				WriteParticle(&going_left, here);
				RESET(going_left);
				// fall through
			case MIDDLE:
				WriteParticle(&going_right, here + 1);
			}
		} else { // if moving right

			ReadParticle(*here, &going_right);

			if (going_right.color) {
				if (going_left.color)
					Collide(&going_left, &going_right);
				if (resident && (going_right.color > resident))
					Bump(&going_right, &resident);
				else if (!resident && !going_left.color && !going_right.momentum)
					Reside(&going_right, &resident);
			}

			__syncthreads();
			// prepare for moving left
			switch (role) {
			case END:
				if (going_right.color)
					DECREASE_MOMENTUM(going_right);
				WriteParticle(&going_right, here);
				RESET(going_right);
				// fall through
			case MIDDLE:
				WriteParticle(&going_left, here - 1);
			}
		}
		if (!resident)
			isNotComplete = TRUE;
		__syncthreads();
	} while (isNotComplete);


	/* read sorted values back to array */
	global_mem[absThreadID] = ((resident - 1) & COLOR_MASK);
}

static __device__ void ReadParticle (const unsigned int src, struct particle *dest)
{
	dest->momentum = src >> COLOR_WIDTH;
	dest->color = src & COLOR_MASK;
}

static __device__ void WriteParticle (const struct particle *src, volatile unsigned int *dest)
{
	*dest = (src->momentum << COLOR_WIDTH) | src->color;
}

static __device__ void Collide (struct particle *L, struct particle *R)
{
	if (L->color < R->color) {
		INCREASE_MOMENTUM_PTR(L);
		INCREASE_MOMENTUM_PTR(R);
	} else {
		DECREASE_MOMENTUM_PTR(L);
		DECREASE_MOMENTUM_PTR(R);
		L->color ^= R->color;
		R->color ^= L->color;
		L->color ^= R->color;
		L->momentum ^= R->momentum;
		R->momentum ^= L->momentum;
		L->momentum ^= R->momentum;
	}
}

static __device__ void Bump (struct particle *incoming, unsigned int *resident)
{
	unsigned int temp = incoming->color;
	incoming->color = *resident;
	DECREASE_MOMENTUM_PTR(incoming);
	*resident = temp;
}

static __device__ void Reside (struct particle *incoming, unsigned int *resident)
{
	*resident = incoming->color;
	incoming->color = 0;
}


/// CUDA HOST /////////////////////////////////////////////////////////////////////////////
static void ErrorCheck (hipError_t cerr, const char *str);
__device__ unsigned int *global_mem;

extern "C" void sort (unsigned int *buffer, unsigned long size)
{
	dim3 grid (1);
	dim3 block (size);
	size_t transfer_size = size * sizeof(int);

	ErrorCheck(hipMalloc(&global_mem, transfer_size), "hipMalloc global");
	ErrorCheck(hipMemcpy(global_mem, buffer, transfer_size, hipMemcpyHostToDevice),
			"hipMemcpy device->host");

	ParticleSort<<<grid, block>>>(global_mem, size);
	hipDeviceSynchronize();
	ErrorCheck(hipGetLastError(), "kernel execution");
	
	ErrorCheck(hipMemcpy(buffer, global_mem, transfer_size, hipMemcpyDeviceToHost),
			"hipMemcpy host->device");
	ErrorCheck(hipFree(global_mem), "hipFree global");
}

static void ErrorCheck (hipError_t cerr, const char *str)
{
	if (cerr == hipSuccess) return;
	fprintf(stderr, "CUDA Runtime Error: %s\n at %s\n", hipGetErrorString(cerr), str);
	exit(EXIT_FAILURE);
}


/// MAIN //////////////////////////////////////////////////////////////////////////////////
int main (int argc, char **argv)
{
	unsigned long elapsed = TestHarness(sort);
	fprintf(stderr, "Sort complete; time elapsed: %lu ms\n", elapsed);
	exit(EXIT_SUCCESS);
}

#endif
