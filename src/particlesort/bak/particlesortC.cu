#include "hip/hip_runtime.h"
#ifndef PARTICLE_SORT_CU
#define PARTICLE_SORT_CU

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../testharness/testharness.h"

#define LIFESPAN 30


enum particleState { ALIVE, DYING, DEAD };


__device__ unsigned short *global_mem;
__device__ int *sums;
__device__ int dead_ct = 0;

__device__ void Collide (signed char *, short *, enum particleState *);
__device__ void Split (signed char *, signed char *, short *, enum particleState *);
__device__ void Pass (short *, enum particleState *);
__device__ void Revive (signed char *, short *, enum particleState *, const signed char *);
__device__ void Die (enum particleState *);


#define COLLIDE Collide(&velocity,&momentum,&state)
#define SPLIT Split(&velocity,&notfirst,&momentum,&state)
#define PASS Pass(&momentum,&state)
#define REVIVE Revive(&velocity,&momentum,&state,&direction)
#define DIE Die(&state)
extern "C" __global__ void ParticleSort (unsigned short *global_mem, int *sums, unsigned long size)
{
	//extern __shared__ int shmem [];
	int *end = sums + size - 1;
	unsigned int threadID = blockDim.x * blockIdx.x + threadIdx.x;

	if (threadID == 0)
		dead_ct = 0;
	signed char direction = -1;

	/* slot initialization */
	int *here = sums + threadID;

	/* particle initialization */
	enum particleState state = ALIVE;
	signed char velocity = 1 - (threadID & 0x01) * 2;
	int *position = here;
	int value = global_mem[threadID] + 1;
	short momentum = LIFESPAN;

	/* main sorting loop */
	do {
		/* prepare */
		*here = 0;
		direction = -direction;
		__syncthreads();

		/* move position if velocity is same as direction */
		/* perform wall collisions */
		if ((state != DEAD) && (velocity == direction)) {
			position += velocity;
			if (position < sums) {
				position = sums;
				COLLIDE;
			} else if (position > end) {
				position = end;
				COLLIDE;
			}
		}

		/* prepare collisions */
		int weighted_value = (state != DEAD) ? velocity : direction;
		weighted_value = (weighted_value > 0) ? value : -value;
		signed char notfirst = atomicAdd(position, weighted_value);
		__syncthreads();
		
		/* resolve collisions */
		int sum = *position;
		int others = abs(sum - weighted_value);
		if (others == 0) {
			if (state == DYING) DIE;
			/* else NOP */
		}
		else if ((sum == 0) && (value != others)) SPLIT; 
		else if (sum < 0) (state != DEAD) ? COLLIDE : REVIVE;
		//else if (sum > 0) PASS;
		/* else NOP */
		__syncthreads();

	} while (dead_ct < size); 
	__syncthreads();

	/* we're done. copy everything back into global memory */
	atomicExch(position, value);
	__syncthreads();
	global_mem[threadID] = (unsigned short)*here - 1;
}

__device__ void Split (signed char *velocity, signed char *notfirst, short *momentum, enum particleState *state)
{
	if (*state == DEAD)
		return;
	*velocity = *notfirst ? 1 : -1;
	if (--(*momentum) <= 0) {
		*state = DYING;
		*momentum = 0;
	}
}

__device__ void Collide (signed char *velocity, short *momentum, enum particleState *state)
{
	*velocity = -(*velocity);
	if (--(*momentum) <= 0) {
		*state = DYING;
		*momentum = 0;
	}
}

__device__ void Pass (short *momentum, enum particleState *state)
{
	if (*state == DEAD)
		return;
	*momentum = min(*momentum + 1, LIFESPAN);
	*state = ALIVE;
}

__device__ void Revive (signed char *velocity, short *momentum, enum particleState *state, const signed char *direction)
{
	*momentum = 1;
	*velocity = *direction;
	*state = DYING;
	atomicAdd(&dead_ct, -1);
}

__device__ void Die (enum particleState *state)
{
	*state = DEAD;
	atomicAdd(&dead_ct, 1);
}



void ErrorCheck (hipError_t cerr, const char *str)
{
	if (cerr == hipSuccess) 
		return;
	fprintf(stderr, "CUDA Runtime Error: %s\n at %s\n", hipGetErrorString(cerr), str);
	exit(-1);
}


extern void sort (unsigned short *buffer, unsigned long size)
{
	ErrorCheck(hipMalloc(&global_mem, size * sizeof(unsigned short)), "hipMalloc global");
	ErrorCheck(hipMalloc(&sums, size * sizeof(int)), "hipMalloc sums");
	
	ErrorCheck(hipMemcpy(global_mem, buffer, size * sizeof(unsigned short), hipMemcpyHostToDevice), "hipMemcpy host->device global");

	dim3 grid (1, 1, 1);
	dim3 block (size, 1, 1);
	/*size_t shmem_size = size * 4;*/
	ParticleSort<<<grid, block>>>(global_mem, sums, size);

	ErrorCheck(hipMemcpy(buffer, global_mem, size * sizeof(unsigned short), hipMemcpyDeviceToHost), "hipMemcpy device->host");

	ErrorCheck(hipFree(global_mem), "hipFree global");
	ErrorCheck(hipFree(sums), "hipFree sums");
}

int main (int argc, char **argv)
{
	unsigned long elapsed = TestHarness(sort);
	fprintf(stderr, "Sort complete; time elapsed: %lu ms\n", elapsed);
	exit(EXIT__SUCCESS);
}

#endif
